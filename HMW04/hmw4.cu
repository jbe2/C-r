
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>



#define PI 3.14159265359

__global__ void solve(const int N, float * u, float *newu, float *f, float *res2v){
	float w = 0.5;
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id>N+2 && id<(N+2)*(N+2)-(N+2) && id%(N+2) !=N+1 && id%(N+2) != 0){
		const float Ru = -u[id-(N+2)] -u[id+(N+2)] - u[id-1] - u[id+1];
		const float rhs = (1./4)*(f[id]-Ru);
		const float unew = w*rhs + (1-w)*u[id];
		newu[id] = unew;
		res2v[id] = (unew-u[id])*(unew-u[id]); 
		}
	}



__global__ void res2fred(int Nblocks, int blockdim, float* res2, float* res2_small){
	int bid = blockIdx.x;
	int I =  blockDim.x*blockIdx.x+threadIdx.x;
	float extra_sum = 0;
	if (bid < Nblocks-1){
		for (int s = blockdim*(Nblocks-1)/2; s>1; s/=2){
				if (I<s){
					res2_small[I] += res2[I+s];
				} 
				__syncthreads();		 		
			}
		}
	else{
		extra_sum += res2[I];
	}
	__syncthreads();
	res2_small[Nblocks-1] = extra_sum;
}	

int main(void)
{
int N = 1000;
double tol = 1e-6;
int Nthreads = 128;
const int Nblocks = (N+Nthreads-1)/Nthreads+1;
dim3 threadsPerblock(Nthreads, 1,1);
dim3 blocks(Nblocks,1,1);
int blockdim = Nthreads;

float *f_c, *u_c, *unew_c, *res2v_c, *res2vsmall_c;
hipMalloc(&u_c,(N+2)*(N+2)*sizeof(float));
hipMalloc(&f_c,(N+2)*(N+2)*sizeof(float));
hipMalloc(&unew_c,(N+2)*(N+2)*sizeof(float));
hipMalloc(&res2v_c, (N+2)*(N+2)*sizeof(float));
hipMalloc(&res2vsmall_c, (N+2)*(N+2)*sizeof(float));

float *u = (float*) calloc((N+2)*(N+2), sizeof(float));
float *f = (float*) calloc((N+2)*(N+2),sizeof(float));
float *unew = (float*) calloc((N+2)*(N+2), sizeof(float));
float *res2v = (float*) calloc((N+2)*(N+2), sizeof(float));
float *res2vsmall = (float*) calloc(100, sizeof(float));
float h = 2.0/(N+1);
	for (int i = 0; i<N+2; i++){
		for (int j = 0; j<N+2; j++){
			const float x = -1 + i*h;
			const float y = -1 + j*h;
			f[i+j*(N+2)] = sin(PI*x)*sin(PI*y)*h*h;
		}
	}
int iter = 0;
float res2 = 1;
float res2sum = 0;
while (res2 > tol*tol){
	hipMemcpy(f_c, f, (N+2)*(N+2)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(u_c, u, (N+2)*(N+2)*sizeof(float), hipMemcpyHostToDevice);
	solve <<<blocks, threadsPerblock>>> (N,u_c,unew_c, f_c, res2v_c);
	res2fred <<<blocks, threadsPerblock >>> (Nblocks, blockdim, res2v_c, res2vsmall_c);	
	hipMemcpy(unew, unew_c, (N+2)*(N+2)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(res2vsmall, res2vsmall_c, 100*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(res2v, res2v_c, (N+2)*(N+2)*sizeof(float), hipMemcpyDeviceToHost);
//	for (int i = 0; i<Nblocks; i++){
//	printf("res2vsmall[%d] = %f\n",i, res2vsmall[i]);	
//	res2sum += res2vsmall[i];
//	}
	for (int i = 0; i<(N+2)*(N+2); i++){
	u[i] = unew[i];
	res2sum +=res2v[i];
	}


	res2 = res2sum;
	res2sum = 0;
//	printf("%d", iter);
	iter++;
}

float err = fabs(u[0]-f[0]/(h*h*2.0*PI*PI));
for (int i = 0; i<(N+2)*(N+2)-1; i++){
	if (err <=fabs(u[i+1]-f[i+1]/(h*h*2.0*PI*PI))){
		err = fabs(u[i+1]-f[i+1]/(h*h*2.0*PI*PI));
		}
	}
printf("iter %d", iter);
hipFree(u_c);
hipFree(f_c);
hipFree(unew_c);
hipFree(res2v_c);

free(u);
free(f);
free(unew);
free(res2v);
return 0;
}

